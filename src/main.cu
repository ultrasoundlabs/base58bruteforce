#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>
#include <limits>
#include <algorithm>

#include "sha256.cuh"

// -----------------------------------------------------------------------------
// Constants
// -----------------------------------------------------------------------------

__constant__ uint8_t kBase58Lookup[128];

constexpr uint8_t INVALID = 0xFF;
constexpr int     BASE58_DECODED_LEN = 25;

// Maximum number of matches we will keep. 1M entries ⇒ 8 MB of GPU memory.
// With 2^32 trials and a 32-bit checksum, the expected number of hits is ~1,
// but we reserve plenty of head-room just in case.
constexpr unsigned int MAX_MATCHES = 1 << 20; // 1'048'576

// The Bitcoin Base58 alphabet (note that 0, O, I, and l are omitted)
static const uint8_t kBase58Alphabet[] =
    "123456789ABCDEFGHJKLMNPQRSTUVWXYZabcdefghijkmnopqrstuvwxyz";

__constant__ char kInputString[64];

// -----------------------------------------------------------------------------
// Error-checking helpers
// -----------------------------------------------------------------------------
#define CUDA_CHECK(expr)                                                     \
    do {                                                                    \
        hipError_t _err = (expr);                                          \
        if (_err != hipSuccess) {                                          \
            fprintf(stderr, "CUDA error %s at %s:%d — %s\n",          \
                    #expr, __FILE__, __LINE__, hipGetErrorString(_err));   \
            std::exit(EXIT_FAILURE);                                        \
        }                                                                   \
    } while (0)

// -----------------------------------------------------------------------------
// Device code
// -----------------------------------------------------------------------------

__device__ __forceinline__ uint8_t lookup_base58(uint8_t c) {
    return (c < 128) ? kBase58Lookup[c] : INVALID;
}

__device__ bool decode_validate_mask(int          in_len,
                                     const int   *letter_idx,
                                     int          num_letters,
                                     uint64_t     mask) {
    constexpr int NUM_LIMBS = 9;
    uint32_t limbs[NUM_LIMBS];
#pragma unroll
    for (int i = 0; i < NUM_LIMBS; ++i) limbs[i] = 0u;

    int letter_pos = 0;

    for (int i = 0; i < in_len; ++i) {
        uint8_t byte = static_cast<uint8_t>(kInputString[i]);

        // Apply case selected by mask
        if ((byte >= 'A' && byte <= 'Z') || (byte >= 'a' && byte <= 'z')) {
            const int bit = static_cast<int>((mask >> letter_pos) & 1ULL);
            ++letter_pos;
            if (bit)
                byte &= ~0x20; // upper
            else
                byte |= 0x20; // lower
        }

        const uint8_t digit = lookup_base58(byte);
        if (digit == INVALID) return false;

        // Multiply the 288-bit integer by 58 and add the new digit.
        uint64_t carry = digit;
#pragma unroll
        for (int l = 0; l < NUM_LIMBS; ++l) {
            uint64_t val = static_cast<uint64_t>(limbs[l]) * 58ULL + carry;
            limbs[l] = static_cast<uint32_t>(val & 0xFFFFFFFFULL);
            carry = val >> 32ULL;
        }
    }

    uint8_t bytes[34];
#pragma unroll
    for (int l = 0; l < NUM_LIMBS; ++l) {
        const uint32_t w = limbs[l];
        bytes[l * 4 + 0] = static_cast<uint8_t>(w & 0xFFu);
        bytes[l * 4 + 1] = static_cast<uint8_t>((w >> 8) & 0xFFu);
        bytes[l * 4 + 2] = static_cast<uint8_t>((w >> 16) & 0xFFu);
        bytes[l * 4 + 3] = static_cast<uint8_t>((w >> 24) & 0xFFu);
    }

    // Find the index of the first non-zero byte starting from MSB side
    int offset = 33;
    while (offset > 0 && bytes[offset] == 0) --offset;

    const int remaining = offset + 1; // bytes 0..offset inclusive
    if (remaining != BASE58_DECODED_LEN) return false;

    if (bytes[offset] != 0x41) return false; // version byte check

    uint8_t payload[BASE58_DECODED_LEN];
#pragma unroll
    for (int i = 0; i < BASE58_DECODED_LEN; ++i)
        payload[i] = bytes[offset - i];

    uint8_t digest[32];
    double_sha256_21(payload, digest);

    // Compare the first 4 bytes of the resulting digest to the checksum in the payload
    return (digest[0] == payload[21] &&
            digest[1] == payload[22] &&
            digest[2] == payload[23] &&
            digest[3] == payload[24]);
}

__global__ void kernel_find_all(int          in_len,
                                const int   *letter_idx,
                                int          num_letters,
                                uint64_t     total_masks,
                                uint64_t    *matches,       // [MAX_MATCHES]
                                unsigned int *match_count)  // single counter
{
    const uint64_t stride = static_cast<uint64_t>(gridDim.x) * blockDim.x;
    uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;

    while (tid < total_masks) {
        if (decode_validate_mask(in_len, letter_idx, num_letters, tid)) {
            // Reserve a slot for this match
            unsigned int idx = atomicAdd(match_count, 1u);
            if (idx < MAX_MATCHES) {
                matches[idx] = tid;
            }
        }
        tid += stride;
    }
}

// -----------------------------------------------------------------------------
// Host helpers
// -----------------------------------------------------------------------------
static std::vector<int> build_letter_index(const std::string &s) {
    std::vector<int> idx;
    for (int i = 0; i < static_cast<int>(s.size()); ++i)
        if (std::isalpha(static_cast<unsigned char>(s[i]))) idx.push_back(i);
    return idx;
}

static void init_lookup_table() {
    uint8_t host[128];
    for (int i = 0; i < 128; ++i) host[i] = INVALID;
    for (int i = 0; i < 58; ++i) {
        host[kBase58Alphabet[i]] = static_cast<uint8_t>(i);
    }
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(kBase58Lookup), host, sizeof(host)));
}

int main(int argc, char **argv) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <ambiguous-base58-string>\n", argv[0]);
        return EXIT_FAILURE;
    }
    std::string input = argv[1];

    if (input.empty() || input.size() > 64) {
        fprintf(stderr, "Input length must be 1..64 characters.\n");
        return EXIT_FAILURE;
    }

    auto letter_idx = build_letter_index(input);
    const int num_letters = static_cast<int>(letter_idx.size());
    if (num_letters == 0) {
        fprintf(stderr, "Input has no ambiguous letters. Nothing to brute-force.\n");
        return EXIT_FAILURE;
    }
    if (num_letters >= 63) {
        fprintf(stderr, "Too many letters (%d) — mask would overflow 64 bits.\n", num_letters);
        return EXIT_FAILURE;
    }

    const uint64_t total_masks = 1ULL << num_letters;
    const int threads_per_block = 256;
    const uint64_t blocks = (std::min<uint64_t>(total_masks, (1ULL << 32))) / threads_per_block + 1;

    // Device allocations
    int     *d_letter_idx = nullptr;
    uint64_t    *d_matches    = nullptr;
    unsigned int *d_match_cnt = nullptr;

    // Copy the input string into constant memory once; all threads will read
    // it from the constant cache.
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(kInputString), input.data(), input.size(), 0, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_letter_idx, letter_idx.size() * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_letter_idx, letter_idx.data(), letter_idx.size() * sizeof(int),
                          hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_matches, MAX_MATCHES * sizeof(uint64_t)));
    CUDA_CHECK(hipMalloc(&d_match_cnt, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(d_match_cnt, 0, sizeof(unsigned int)));

    init_lookup_table();

    kernel_find_all<<<static_cast<uint32_t>(blocks), threads_per_block>>>(
        static_cast<int>(input.size()), d_letter_idx, num_letters,
        total_masks, d_matches, d_match_cnt);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    unsigned int host_match_cnt = 0;
    CUDA_CHECK(hipMemcpy(&host_match_cnt, d_match_cnt, sizeof(unsigned int), hipMemcpyDeviceToHost));

    std::vector<uint64_t> host_matches(host_match_cnt);
    if (host_match_cnt > 0) {
        CUDA_CHECK(hipMemcpy(host_matches.data(), d_matches,
                              host_match_cnt * sizeof(uint64_t),
                              hipMemcpyDeviceToHost));
        std::sort(host_matches.begin(), host_matches.end());
    }

    CUDA_CHECK(hipFree(d_letter_idx));
    CUDA_CHECK(hipFree(d_matches));
    CUDA_CHECK(hipFree(d_match_cnt));

    if (host_match_cnt == 0) {
        fprintf(stderr, "No valid candidate found\n");
        return EXIT_FAILURE;
    }

    for (uint64_t mask : host_matches) {
        std::string corrected = input;
        int idx = 0;
        for (size_t i = 0; i < corrected.size(); ++i) {
            if (std::isalpha(static_cast<unsigned char>(corrected[i]))) {
                const int bit = (mask >> idx) & 1ULL;
                corrected[i] = bit ? static_cast<char>(std::toupper(corrected[i]))
                                    : static_cast<char>(std::tolower(corrected[i]));
                ++idx;
            }
        }
        printf("%s\n", corrected.c_str());
    }
    return EXIT_SUCCESS;
} 